#include "hip/hip_runtime.h"

__global__ void init_kernel(int * domain, int domain_x)
{
	// Dummy initialization
	domain[blockIdx.y * domain_x + blockIdx.x * blockDim.x + threadIdx.x]
		= (1664525ul * (blockIdx.x + threadIdx.y + threadIdx.x) + 1013904223ul) % 3;
}

// Reads a cell at (x+dx, y+dy)
__device__ int read_cell(int * source_domain, int x, int y, int dx, int dy,
    unsigned int domain_x, unsigned int domain_y)
{
    x = (unsigned int)(x + dx) % domain_x;	// Wrap around
    y = (unsigned int)(y + dy) % domain_y;
    return source_domain[y * domain_x + x];
}

// Compute kernel
__global__ void life_kernel(int * source_domain, int * dest_domain,
    int domain_x, int domain_y, int shared_x, int shared_y)
{

    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y;

    // Shared memory
    extern __shared__ int block_row[];

    block_row[threadIdx.y * shared_x + threadIdx.x] = source_domain[];
    __syncthreads();


    // Read cell
    int myself = read_cell(block_row, tx, ty, 0, 0,
	                       domain_x, domain_y);


    //Read the 8 neighbors and count number of blue and red
    int i, j, tmp, value, nbRed = 0, nbBlue = 0;
    for (i = -1 ; i <= 1 ; i++){
        for (j = -1 ; j <= 1 ; j++){
            if(i!=0 || j!=0){
                tmp = read_cell( block_row, tx, ty, i, j, domain_x, domain_y);
                switch(tmp){
                    case 1:{
                        nbRed++;
                        break;
                    }
                    case 2:{
                        nbBlue++;
                        break;
                    }
                }
            }
        }
    }

	//Compute new value

    //DIE
    if ((nbBlue+nbRed<2) || (nbBlue+nbRed>3)) {
        value = 0;
    }

    // New cell
    else if ((myself == 0) && (nbBlue+nbRed==3)) {

        value = (nbBlue > nbRed) ? 2 : 1;

    }

	//Write it in dest_domain
    dest_domain[ty * domain_x + tx] = value;
}
