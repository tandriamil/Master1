
#include "utils.h"
#include <stdlib.h>

#include "life_kernel.cu"

void init_data(int * domain, int domain_x, int domain_y)
{
	for(int i = 0; i != domain_y; ++i) {
		for(int j = 0; j != domain_x; ++j) {
			domain[i * domain_x + j] = rand() % 3;
		}
	}
}

int main(int argc, char ** argv)
{
    // Definition of parameters
    int domain_x = 128;	// Multiple of threads_per_block * cells_per_word
    int domain_y = 128;

    int cells_per_word = 1;

    int steps = 2;

    int threads_per_block = 128;
    int blocks_x = domain_x / (threads_per_block * cells_per_word);
    int blocks_y = domain_y;

    dim3  grid(blocks_x, blocks_y);	// CUDA grid dimensions
    dim3  threads(threads_per_block);	// CUDA block dimensions

    // Allocation of arrays
    int * domain_gpu[2] = {NULL, NULL};

	// Arrays of dimensions domain.x * domain.y
	size_t domain_size = domain_x * domain_y / cells_per_word * sizeof(int);
	CUDA_SAFE_CALL(hipMalloc((void**)&domain_gpu[0], domain_size));
    CUDA_SAFE_CALL(hipMalloc((void**)&domain_gpu[1], domain_size));

    int * domain_cpu = (int*)malloc(domain_size);

	// Arrays of dimensions pitch * domain.y

	init_data(domain_cpu, domain_x, domain_y);
    CUDA_SAFE_CALL(hipMemcpy(domain_gpu[0], domain_cpu, domain_size, hipMemcpyHostToDevice));
	//init_kernel<<< grid, threads, 0 >>>(domain_gpu[0], domain_x);

    // Timer initialization
    hipEvent_t start, stop;
    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&stop));

    // Start timer
    CUDA_SAFE_CALL(hipEventRecord(start, 0));

    // Get the sizes
    int shared_x = (domain_x / blocks_x) + 2;
    int shared_y = (domain_y / blocks_y) + 2;

    // Kernel execution
    int shared_mem_size =  shared_x * shared_y * sizeof(int);
    for(int i = 0; i < steps; i++) {
	    life_kernel<<< grid, threads, shared_mem_size >>>(domain_gpu[i%2],
	    	domain_gpu[(i+1)%2], domain_x, domain_y, (domain_x / blocks_x), (domain_y / blocks_y));
	}

    // Stop timer
    CUDA_SAFE_CALL(hipEventRecord(stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(stop));

    float elapsedTime;
    CUDA_SAFE_CALL(hipEventElapsedTime(&elapsedTime, start, stop));	// In ms
    printf("GPU time: %f ms\n", elapsedTime);

    CUDA_SAFE_CALL(hipEventDestroy(start));
    CUDA_SAFE_CALL(hipEventDestroy(stop));

    // Get results back
    CUDA_SAFE_CALL(hipMemcpy(domain_cpu, domain_gpu[steps%2], domain_size, hipMemcpyDeviceToHost));

    CUDA_SAFE_CALL(hipFree(domain_gpu[0]));
    CUDA_SAFE_CALL(hipFree(domain_gpu[1]));


    // Count colors
    int red = 0;
    int blue = 0;
    for(int y = 0; y < domain_y; y++)
    {
    	for(int x = 0; x < domain_x; x++)
    	{
    		int cell = domain_cpu[y * domain_x + x];
    		printf("%u", cell);
    		if(cell == 1) {
    			red++;
    		}
    		else if(cell == 2) {
    			blue++;
    		}
    	}
    	printf("\n");
    }

    printf("Red/Blue cells: %d/%d\n", red, blue);

    free(domain_cpu);

    return 0;
}
