
#include <hip/hip_runtime.h>
// GPU kernel
__global__ void summation_kernel(int data_size, float * data_out) {

	// Get the id of this thread in the whole thread group
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;

	// Get the total number of threads in the whole thread group
	unsigned int nb_threads_total = blockDim.x * gridDim.x;

	// Get the number of part for each thread
	unsigned int units_per_thread = nb_threads_total / data_size;

	// Compute datas for this thread
	int i;
	float res = 0;
	for (i = ((id+1) * units_per_thread)-1; i >= (id * units_per_thread); --i) {
		if (i%2==0) res += 1.0/(i+1.0);
		else res -= 1.0/(i+1.0);
	}

	// Store the result
	data_out[id] = res;
}
