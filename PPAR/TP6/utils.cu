// Fonctions auxiliaires
#ifdef _WIN32
#define WIN32_LEAN_AND_MEAN
#define NOMINMAX
#include <windows.h>
#else

#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <time.h>
#endif


/**
 * Generate Random number in single precision.
 */
float random_float(int emin, int emax, int pos_neg) {
	double tmp;
	unsigned int i, val;
	int e;

	val = (rand() & 0x000000ff);
	for (i = 0; i < sizeof(int); ++i) {
		val = val << 8;
		val += (rand() & 0x000000ff); /* we keep only 8 bits */
	}
	e = emin + (int)((double)rand()*(emax-emin)/(double)RAND_MAX);
	tmp = ldexp(1.0 + (double)val / UINT_MAX, e);
	if ((pos_neg) && (rand() > (RAND_MAX/2))) tmp *= -1;

	return (float)tmp;
}


double getclock() {
	#ifdef _WIN32
		LARGE_INTEGER li;
		QueryPerformanceFrequency(&li);

		double PCFreq = (double)li.QuadPart;
		QueryPerformanceCounter(&li);
		__int64 timerStart = li.QuadPart;
		return ((double)li.QuadPart)/PCFreq;
	#else
		struct timeval tv;
		gettimeofday(&tv, NULL);
		return tv.tv_sec + double(tv.tv_usec) / 1000000.;
	#endif
}
