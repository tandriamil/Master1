#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdlib.h>

struct results {
	float sum;
};

#include "summation_kernel.cu"

// CPU implementation
float log2_series(int n) {
	int i;
	float res = 0;
	for (i = n - 1; i >= 0; --i)
	{
		if(i%2==0) res += 1.0/(i+1.0);
		else res -= 1.0/(i+1.0);
	}
	return res;
}


int main(int argc, char ** argv) {
	int data_size = 1024 * 1024 * 128;

	// Run CPU version
	double start_time = getclock();
	float log2 = log2_series(data_size);
	double end_time = getclock();

	printf("CPU result: %f\n", log2);
	printf(" log(2)=%f\n", log(2.0));
	printf(" time=%fs\n", end_time - start_time);

	// Parameter definition
	int threads_per_block = 4 * 32;
	int blocks_in_grid = 8;

	int num_threads = threads_per_block * blocks_in_grid;

	// Timer initialization
	hipEvent_t start, stop;
	CUDA_SAFE_CALL(hipEventCreate(&start));
	CUDA_SAFE_CALL(hipEventCreate(&stop));

	int results_size = num_threads;


	// As the input is just an int, we don't have to allocate GPU memory for it
	// We'll just pass it as parameter

	// Allocating output data on CPU
	float data_out_cpu[results_size];

	// Allocating output data on GPU
	float * data_out_gpu;
	hipMalloc((void **)&data_out_gpu, results_size*sizeof(float));

	// Start timer
	CUDA_SAFE_CALL(hipEventRecord(start, 0));

	// Launch kernel
	summation_kernel<<<blocks_in_grid, threads_per_block>>>(data_size, data_out_gpu);

	// Stop timer
	CUDA_SAFE_CALL(hipEventRecord(stop, 0));
	CUDA_SAFE_CALL(hipEventSynchronize(stop));

	// Get results back
	hipMemcpy(data_out_cpu, data_out_gpu, results_size*sizeof(float), hipMemcpyDeviceToHost);


	// Finish reduction
	float sum = 0.;
	int i, units_per_thread = data_size / num_threads;
	for (i = 0; i < results_size; ++i) {
		if((i*units_per_thread)%2==0) res += data_out_cpu[i];
		else res -= data_out_cpu[i];
	}

	// Cleanup
	hipFree(data_out_gpu);


	printf("GPU results:\n");
	printf(" Sum: %f\n", sum);

	float elapsedTime;
	CUDA_SAFE_CALL(hipEventElapsedTime(&elapsedTime, start, stop));	// In ms

	double total_time = elapsedTime / 1000.;	// s
	double time_per_iter = total_time / (double)data_size;
	double bandwidth = sizeof(float) / time_per_iter; // B/s

	printf(" Total time: %g s,\n Per iteration: %g ns\n Throughput: %g GB/s\n",
		total_time,
		time_per_iter * 1.e9,
		bandwidth / 1.e9);

	CUDA_SAFE_CALL(hipEventDestroy(start));
	CUDA_SAFE_CALL(hipEventDestroy(stop));
	return 0;
}
