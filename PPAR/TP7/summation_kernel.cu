
#include <hip/hip_runtime.h>

// GPU kernel
__global__ void summation_kernel(int data_size, float * data_out)
{
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int nb_threads_total = blockDim.x * gridDim.x;
	unsigned int units_per_thread = data_size / nb_threads_total;
	int i;
	float res = 0.;

	printf("Before launching the GPU thread id:%d nb threads:%d units per thread:%d \n", id, nb_threads_total, units_per_thread);
	for (i = ((id+1) * units_per_thread)-1; i >= int(id * units_per_thread); i--) {
		if(id==0)printf("boucle:%d",i);
        if(i%2==0) res += 1.0/(i+1.0);
        else res -= 1.0/(i+1.0);
	}
	printf("Getting the result for thread %d which is %f \n", id, res);
	data_out[id] = res;
	printf("Thread %d has ended\n", id);
}
